#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string.h>
#include <time.h>
#include "Graph.h"

#define BLOCK_SIZE_1D 256
#define BLOCK_SIZE_2D 16

using namespace std;

//******************************************************************************
// Kernels to update the Matrix at k-th iteration

__global__ void floyd_kernel1D(int * M, const int nverts, const int k) {
  int ij = blockIdx.x * blockDim.x + threadIdx.x,
      i = ij / nverts,
      j = ij - i * nverts;
  if (i < nverts && j < nverts) {
    if (i != j && i != k && j != k) {
      M[ij] = min(M[i * nverts + k]  + M[k * nverts + j], M[ij]);
    }
  }
}

__global__ void floyd_kernel2D(int * M, const int nverts, const int k) {
  int ii = blockIdx.y * blockDim.y + threadIdx.y,
      jj = blockIdx.x * blockDim.x + threadIdx.x,
      i = ii / nverts,
      j = jj / nverts;
  if (i < nverts && j < nverts) {
    if (i != j && i != k && j != k) {
      int ij = i * nverts + j,
          ik = i * nverts + k,
          kj = k * nverts + j;
      M[ij] = min(M[ik] + M[kj], M[ij]);
    }
  }
}


//******************************************************************************
// Main

int main(int argc, char *argv[]) {
  if (argc != 2) {
    cerr << "Sintaxis: " << argv[0] << " <archivo de grafo>" << endl;
    return(-1);
	}

  // Get GPU information
  int devID;
  hipDeviceProp_t props;
  hipError_t err;
  err = hipGetDevice(&devID);
  if (err != hipSuccess) {
    cout << "ERRORRR" << endl;
    return(-1);
  }
  hipGetDeviceProperties(&props, devID);
  printf("Device %d: \"%s\" with Compute %d.%d capability\n",
        devID, props.name, props.major, props.minor);

  //****************************************************************************

  Graph G;
  G.lee(argv[1]); // Read the Graph
  //cout << "El Grafo de entrada es:" << endl;
  //G.imprime();

  const int nverts = G.vertices;                              // Vertices
  const int niters = nverts;                                  // Iteraciones
  const int nverts2 = nverts * nverts;                        // Elementos

  const dim3 blocksize1D (BLOCK_SIZE_1D);                     // Tama Bloque 1D
  const dim3 blocksize2D (BLOCK_SIZE_2D, BLOCK_SIZE_2D);      // Tama Bloque 2D
  const dim3 nblocks1D (ceil((float) (nverts * nverts) / blocksize1D.x)); // Bloques 1D
  const dim3 nblocks2D (ceil((float) nverts / blocksize2D.x), // Bloques 2D
                        ceil((float) nverts / blocksize2D.y));

  int * c_out_M_1D = new int[nverts2];  // Matriz en el HOST 1D
  int * c_out_M_2D = new int[nverts2];  // Matriz en el HOST 2D
  int size = nverts2 * sizeof(int);     // Tama en bytes de la matriz de salida
  int * d_In_M_1D = NULL;               // Matriz en DEVICE para 1D
  int * d_In_M_2D = NULL;               // Matriz en DEVICE para 2D

  int i, j, k;
  double T, Tgpu1D, Tgpu2D, Tcpu;

  //****************************************************************************
  // GPU phase (1D)

  // Reservar espacio en memoria para la matriz en DEVICE
  err = hipMalloc((void **) &d_In_M_1D, size);
  if (err != hipSuccess) {
    cout << "ERROR: Bad Allocation in Device Memory" << endl;
  }

  T = clock();

  // Copiar los datos de la matriz en HOST en la matriz en DEVICE
  err = hipMemcpy(d_In_M_1D, G.Get_Matrix(), size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "ERROR: COPY MATRIX TO DEVICE" << endl;
  }

  for (k = 0; k < niters; k++) {
    // Kernel Launch
    floyd_kernel1D <<< nblocks1D, blocksize1D >>> (d_In_M_1D, nverts, k);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel!\n");
      exit(EXIT_FAILURE);
    }
  }

  // Copiar los datos de la matriz en DEVICE en la matriz en HOST
  hipMemcpy(c_out_M_1D, d_In_M_1D, size, hipMemcpyDeviceToHost);

  Tgpu1D = clock();
  Tgpu1D = (Tgpu1D - T) / CLOCKS_PER_SEC;
  cout << "Tiempo gastado GPU (1D) = " << Tgpu1D << endl;

  //****************************************************************************
  // GPU phase (2D)

  // Reservar espacio en memoria para la matriz en DEVICE
  err = hipMalloc((void **) &d_In_M_2D, size);
  if (err != hipSuccess) {
    cout << "ERROR: Bad Allocation in Device Memory" << endl;
  }

  T = clock();

  // Copiar los datos de la matriz en HOST en la matriz en DEVICE
  err = hipMemcpy(d_In_M_2D, G.Get_Matrix(), size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "ERROR: COPY MATRIX TO DEVICE" << endl;
  }

  for (k = 0; k < niters; k++) {
    // Kernel Launch
    floyd_kernel2D <<< nblocks2D, blocksize2D >>> (d_In_M_2D, nverts, k);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel!\n");
      exit(EXIT_FAILURE);
    }
  }

  // Copiar los datos de la matriz en DEVICE en la matriz en HOST
  hipMemcpy(c_out_M_2D, d_In_M_2D, size, hipMemcpyDeviceToHost);

  Tgpu2D = clock();
  Tgpu2D = (Tgpu2D - T) / CLOCKS_PER_SEC;
  cout << "Tiempo gastado GPU (2D) = " << Tgpu2D << endl;

  //****************************************************************************
  // CPU phase

  T = clock();
  // Bucle ppal del algoritmo
  for (int k = 0; k < niters; k++)
    for (int i = 0; i < nverts; i++)
      for (int j = 0; j < nverts; j++)
        if (i != j && i != k && j != k) {
          int vikj = min(G.arista(i, k) + G.arista(k, j), G.arista(i, j));
          G.inserta_arista(i, j, vikj);
        }

  Tcpu = clock();
  Tcpu = (Tcpu - T) / CLOCKS_PER_SEC;
  //cout << endl << "El Grafo con las distancias de los caminos más cortos es:"
  //     << endl << endl;
  //G.imprime();
  cout << "Tiempo gastado CPU = " << Tcpu << endl;
  cout << "Ganancia (1D) = " << Tcpu / Tgpu1D << endl;
  cout << "Ganancia (2D) = " << Tcpu / Tgpu2D << endl;

  //****************************************************************************

  // Comprobar que los resultados en CPU y GPU son los mismos
  for (i = 0; i < nverts; i++)
    for (j = 0; j < nverts; j++) {
      if (abs(c_out_M_1D[i * nverts + j] - G.arista(i, j)) > 0)
        cout << "Error 1D (" << i << "," << j << ")   "
             << c_out_M_1D[i * nverts + j] << "..." << G.arista(i, j) << endl;
      if (abs(c_out_M_2D[i * nverts + j] - G.arista(i, j)) > 0)
        cout << "Error 2D (" << i << "," << j << ")   "
             << c_out_M_2D[i * nverts + j] << "..." << G.arista(i, j) << endl;
    }

  // Liberar memoria
  hipFree(d_In_M_1D);
  hipFree(d_In_M_2D);
  delete[] c_out_M_1D;
  delete[] c_out_M_2D;
}
