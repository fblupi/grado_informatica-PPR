#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string.h>
#include <time.h>
#include "Graph.h"

//#define TIEMPOS // Comentar para obtener resultados de la CPU y comparar con estos los de la GPU

#define BLOCK_SIZE_1D 256
#define BLOCK_SIZE_2D 16

using namespace std;

//**************************************************************************************************
// Kernels to update the Matrix at k-th iteration

// Kernel 1D
__global__ void floyd_kernel1D(int * M, const int nverts, const int k) {
  int ij = blockIdx.x * blockDim.x + threadIdx.x,
      i = ij / nverts,
      j = ij - i * nverts;
  if (i < nverts && j < nverts) {
    if (i != j && i != k && j != k) {
      M[ij] = min(M[i * nverts + k]  + M[k * nverts + j], M[ij]);
    }
  }
}

// Kernel 2D
__global__ void floyd_kernel2D(int * M, const int nverts, const int k) {
  int ii = blockIdx.y * blockDim.y + threadIdx.y,
      jj = blockIdx.x * blockDim.x + threadIdx.x,
      ij = ii * nverts + jj,
      i = ij / nverts,
      j = ij - i * nverts;
  if (i < nverts && j < nverts) {
    if (i != j && i != k && j != k) {
      M[ij] = min(M[i * nverts + k] + M[k * nverts + j], M[ij]);
    }
  }
}

// Kernel Shared 1D
__global__ void floyd_kernel1DShared(int * d_M, const int nverts, const int k) {
  int blockPos = blockIdx.x * blockDim.x, // posicion inicial del bloque en memoria
      g_ij = blockPos + threadIdx.x,      // índice global de memoria == ij
      l_ij = threadIdx.x,                 // índice local en el vector de memoria compartida
      i = g_ij / nverts,                  // índice i en la matriz
      j = g_ij - i * nverts,              // índice j en la matriz
      g_kj = k * nverts + j,              // celda (k, j) en el vector en DEVICE
      l_kj = l_ij + BLOCK_SIZE_1D,        // celda (k, j) en el vector en SHARED
      g_ik = i * nverts + k,              // celda (i, k) en el vector en DEVICE
      l_ik = 2 * BLOCK_SIZE_1D,           // celda (i, k) en el vector en SHARED
      l_i1k = 2 * BLOCK_SIZE_1D + 1,      // celda (i + 1, k) en el vector en SHARED
      blockRow = floor((float) blockPos / nverts),  // fila de la primera hebra del bloque
      threadRow = floor((float) g_ij / nverts);     // fila de la hebra

  __shared__ int s_M[2 * BLOCK_SIZE_1D + 2];
  s_M[l_ij] = d_M[g_ij];    // Copia la celda correspondiente a la fila i
  s_M[l_kj] = d_M[g_kj];    // Copia la celda correspondiente a la fila j
  if (blockRow == threadRow) {
    s_M[l_ik] = d_M[g_ik];  // Copia la celda (i, k)
  } else {
    s_M[l_i1k] = d_M[g_ik]; // Copia la celda (i + 1, k)
  }
  __syncthreads();

  if (i < nverts && j < nverts) {
    //printf("(i=%u, j=%u) => %u..%u\n", i,j,blockRow,threadRow);
    if (i != j && i != k && j != k) {
      if (blockRow == threadRow) {
        if (d_M[g_ij] != s_M[l_ij] ||  d_M[g_kj] != s_M[l_kj] || d_M[g_ik] != s_M[l_ik])
          printf("(i=%u, j=%u, k=%u) => %u\n\t[ij] => d_M=%u...s_M=%u\n\t[kj] => d_M=%u...s_M=%u\n\t[ik] => d_M=%u...s_M=%u\n\n",
            i, j, k, l_ij, d_M[g_ij], s_M[l_ij], d_M[g_kj], s_M[l_kj], d_M[g_ik], s_M[l_ik]);
        d_M[g_ij] = min(s_M[l_ik] + s_M[l_kj], s_M[l_ij]);
      } else {
        if (d_M[g_ij] != s_M[l_ij] ||  d_M[g_kj] != s_M[l_kj] || d_M[g_ik] != s_M[l_i1k])
          printf("(i=%u, j=%u, k=%u) => %u\n\t[ij] => d_M=%u...s_M=%u\n\t[kj] => d_M=%u...s_M=%u\n\t[ik] => d_M=%u...s_M=%u\n\n",
            i, j, k, l_ij, d_M[g_ij], s_M[l_ij], d_M[g_kj], s_M[l_kj], d_M[g_ik], s_M[l_i1k]);
        d_M[g_ij] = min(s_M[l_i1k] + s_M[l_kj], s_M[l_ij]);
      }
    }
  }
}


//**************************************************************************************************
// Main

int main(int argc, char *argv[]) {
  if (argc != 2) {
    cerr << "Sintaxis: " << argv[0] << " <archivo de grafo>" << endl;
    return(-1);
	}

  // Get GPU information
  int devID;
  hipDeviceProp_t props;
  hipError_t err;
  err = hipGetDevice(&devID);
  if (err != hipSuccess) {
    cout << "ERRORRR" << endl;
    return(-1);
  }
  hipGetDeviceProperties(&props, devID);
  printf("Device %d: \"%s\" with Compute %d.%d capability\n",
        devID, props.name, props.major, props.minor);

//**************************************************************************************************

  Graph G;
  G.lee(argv[1]); // Read the Graph
  //cout << "El Grafo de entrada es:" << endl;
  //G.imprime();

  const int nverts = G.vertices;        // Vertices
  const int niters = nverts;            // Iteraciones
  const int nverts2 = nverts * nverts;  // Elementos

  const dim3 blocksize1D (BLOCK_SIZE_1D);                                 // Tama Bloque 1D
  const dim3 blocksize2D (BLOCK_SIZE_2D, BLOCK_SIZE_2D);                  // Tama Bloque 2D
  const dim3 nblocks1D (ceil((float) (nverts * nverts) / blocksize1D.x)); // Num Bloques 1D
  const dim3 nblocks2D (ceil((float) nverts / blocksize2D.x),             // Num Bloques 2D
                        ceil((float) nverts / blocksize2D.y));

  int * c_out_M_1D = new int[nverts2];        // Matriz en el HOST 1D
  int * c_out_M_2D = new int[nverts2];        // Matriz en el HOST 2D
  int * c_out_M_1DShared = new int[nverts2];  // Matriz en el HOST 1D Shared
  int size = nverts2 * sizeof(int);           // Tama en bytes de la matriz de salida
  int * d_In_M_1D = NULL;                     // Matriz en DEVICE para 1D
  int * d_In_M_2D = NULL;                     // Matriz en DEVICE para 2D
  int * d_In_M_1DShared = NULL;               // Matriz en DEVICE para 2D

  int i, j, k;
  double T, Tgpu1D, Tgpu2D, Tgpu1DShared, Tcpu;

  //************************************************************************************************
  // GPU phase (1D)

  // Reservar espacio en memoria para la matriz en DEVICE
  err = hipMalloc((void **) &d_In_M_1D, size);
  if (err != hipSuccess) {
    cout << "ERROR: Bad Allocation in Device Memory" << endl;
  }

  T = clock();

  // Copiar los datos de la matriz en HOST en la matriz en DEVICE
  err = hipMemcpy(d_In_M_1D, G.Get_Matrix(), size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "ERROR: COPY MATRIX TO DEVICE" << endl;
  }

  for (k = 0; k < niters; k++) {
    // Kernel Launch
    floyd_kernel1D <<< nblocks1D, blocksize1D >>> (d_In_M_1D, nverts, k);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel!\n");
      exit(EXIT_FAILURE);
    }
  }

  // Copiar los datos de la matriz en DEVICE en la matriz en HOST
  hipMemcpy(c_out_M_1D, d_In_M_1D, size, hipMemcpyDeviceToHost);

  Tgpu1D = clock();
  Tgpu1D = (Tgpu1D - T) / CLOCKS_PER_SEC;
  cout << "Tiempo gastado GPU (1D) = " << Tgpu1D << endl;

  //************************************************************************************************
  // GPU phase (2D)

  // Reservar espacio en memoria para la matriz en DEVICE
  err = hipMalloc((void **) &d_In_M_2D, size);
  if (err != hipSuccess) {
    cout << "ERROR: Bad Allocation in Device Memory" << endl;
  }

  T = clock();

  // Copiar los datos de la matriz en HOST en la matriz en DEVICE
  err = hipMemcpy(d_In_M_2D, G.Get_Matrix(), size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "ERROR: COPY MATRIX TO DEVICE" << endl;
  }

  for (k = 0; k < niters; k++) {
    // Kernel Launch
    floyd_kernel2D <<< nblocks2D, blocksize2D >>> (d_In_M_2D, nverts, k);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel!\n");
      exit(EXIT_FAILURE);
    }
  }

  // Copiar los datos de la matriz en DEVICE en la matriz en HOST
  hipMemcpy(c_out_M_2D, d_In_M_2D, size, hipMemcpyDeviceToHost);

  Tgpu2D = clock();
  Tgpu2D = (Tgpu2D - T) / CLOCKS_PER_SEC;
  cout << "Tiempo gastado GPU (2D) = " << Tgpu2D << endl;

  //************************************************************************************************
  // GPU phase (1D Shared Memory)

  // Reservar espacio en memoria para la matriz en DEVICE
  err = hipMalloc((void **) &d_In_M_1DShared, size);
  if (err != hipSuccess) {
    cout << "ERROR: Bad Allocation in Device Memory" << endl;
  }

  T = clock();

  // Copiar los datos de la matriz en HOST en la matriz en DEVICE
  err = hipMemcpy(d_In_M_1DShared, G.Get_Matrix(), size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "ERROR: COPY MATRIX TO DEVICE" << endl;
  }

  for (k = 0; k < niters; k++) {
    // Kernel Launch
    floyd_kernel1DShared <<< nblocks1D, blocksize1D >>> (d_In_M_1DShared, nverts, k);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel!\n");
      exit(EXIT_FAILURE);
    }
  }

  // Copiar los datos de la matriz en DEVICE en la matriz en HOST
  hipMemcpy(c_out_M_1DShared, d_In_M_1DShared, size, hipMemcpyDeviceToHost);

  Tgpu1DShared = clock();
  Tgpu1DShared = (Tgpu1DShared - T) / CLOCKS_PER_SEC;
  cout << "Tiempo gastado GPU (1D Shared) = " << Tgpu1DShared << endl;

#ifndef TIEMPOS
  //************************************************************************************************
  // CPU phase

  T = clock();
  // Bucle ppal del algoritmo
  for (k = 0; k < niters; k++) {
    for (i = 0; i < nverts; i++) {
      for (j = 0; j < nverts; j++) {
        if (i != j && i != k && j != k) {
          int vikj = min(G.arista(i, k) + G.arista(k, j), G.arista(i, j));
          G.inserta_arista(i, j, vikj);
        }
      }
    }
  }

  Tcpu = clock();
  Tcpu = (Tcpu - T) / CLOCKS_PER_SEC;
  //cout << endl << "El Grafo con las distancias de los caminos más cortos es:"
  //     << endl << endl;
  //G.imprime();
  cout << "Tiempo gastado CPU = " << Tcpu << endl;
  cout << "Ganancia (1D) = " << Tcpu / Tgpu1D << endl;
  cout << "Ganancia (2D) = " << Tcpu / Tgpu2D << endl;
  cout << "Ganancia (1D Shared) = " << Tcpu / Tgpu1DShared << endl;

  //************************************************************************************************
  // Comprobar que los resultados en CPU y GPU son los mismos

  for (i = 0; i < nverts; i++) {
    for (j = 0; j < nverts; j++) {
      if (abs(c_out_M_1D[i * nverts + j] - G.arista(i, j)) > 0) {
        cout << "Error 1D (" << i << "," << j << ")   " << c_out_M_1D[i * nverts + j]
             << "..." << G.arista(i, j) << endl;
      }
      if (abs(c_out_M_2D[i * nverts + j] - G.arista(i, j)) > 0) {
        cout << "Error 2D (" << i << "," << j << ")   " << c_out_M_2D[i * nverts + j]
             << "..." << G.arista(i, j) << endl;
      }
      if (abs(c_out_M_1DShared[i * nverts + j] - G.arista(i, j)) > 0) {
        cout << "Error 1D Shared (" << i << "," << j << ")   " << c_out_M_1DShared[i * nverts + j]
             << "..." << G.arista(i, j) << endl;
      }
    }
  }
  //************************************************************************************************
#endif

  // Liberar memoria
  hipFree(d_In_M_1D);
  hipFree(d_In_M_2D);
  hipFree(d_In_M_1DShared);
  delete[] c_out_M_1D;
  delete[] c_out_M_2D;
  delete[] c_out_M_1DShared;
}
